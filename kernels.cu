#include "hip/hip_runtime.h"
#include "arena.h"
#include "br_asm.h"
#include "cpu_util.h"
#include "expand.h"
#include "timer.h"
#include "util.h"
#include <stdio.h>
#include <cuda/std/atomic>

#ifdef __i386__
#define MAX_PARALLEL (6)  // maximum number of chases in parallel
#else
#define MAX_PARALLEL (10)
#endif

typedef struct chase_t chase_t;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef union {
  char pad[AVOID_FALSE_SHARING];
  struct {
    unsigned thread_num;        // which thread is this
    unsigned count;             // count of number of iterations
    void *cycle[MAX_PARALLEL];  // initial address for the chases
    const char *extra_args;
    int dummy;  // useful for confusing the compiler

    const struct generate_chase_common_args *genchase_args;
    size_t nr_threads;
    const chase_t *chase;
    void *flush_arena;
    size_t cache_flush_size;
    bool use_longer_chase;
    int branch_chunk_size;
  } x;
} per_thread_t;

__global__ void chase_simple_kernel(per_thread_t *t) {
  void *p = t->x.cycle[0];

  do {
    x200(p = *(void **)p;)
//   } while (__sync_add_and_fetch(&t->x.count, 200));
  } while (((cuda::std::atomic<unsigned> *)&t->x.count)->fetch_add(200) + 200);
  printf("CIAOOOOOO\n");
  // we never actually reach here, but the compiler doesn't know that
  t->x.dummy = (uintptr_t)p;
}

extern "C" {
    void chase_simple_kernel_gpu(per_thread_t *t) {
        chase_simple_kernel<<<1, 1>>>(t);
        hipDeviceSynchronize();
    }
}